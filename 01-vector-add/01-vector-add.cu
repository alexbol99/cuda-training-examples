
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  
  for(int i = idx; i < N; i+=stride)
  {
    result[i] = a[i] + b[i];
  }
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

#define CUDA_STATUS_CHECK(_err) { if (_err != hipSuccess) printf("Cuda error: %s\n", hipGetErrorString(_err)); }

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);
  hipError_t err;
  int threads_per_block = 256;
  int num_blocks = 40;
  
  float *a;
  float *b;
  float *c;

  err = hipMallocManaged(&a, size);
  CUDA_STATUS_CHECK(err);
  err = hipMallocManaged(&b, size);
  CUDA_STATUS_CHECK(err);
  err = hipMallocManaged(&c, size);
  CUDA_STATUS_CHECK(err);
  
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<num_blocks,threads_per_block>>>(c, a, b, N);
  err = hipGetLastError();
  CUDA_STATUS_CHECK(err);
  
  err = hipDeviceSynchronize();
  CUDA_STATUS_CHECK(err);
  
  checkElementsAre(7, c, N);

  err = hipFree(a);
  CUDA_STATUS_CHECK(err);
  err = hipFree(b);
  CUDA_STATUS_CHECK(err);
  err = hipFree(c);
  CUDA_STATUS_CHECK(err);
}
